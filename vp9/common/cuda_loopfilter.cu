#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "cuda_loopfilter_def.h"
#include "cuda_loopfilter.h"
#include "cuda_loopfilters.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__forceinline__ __device__
void filter_cols(const LOOP_FILTER_MASK* const lfm,
                 const cuda_loop_filter_thresh* const lft,
                 const uint32_t sx,
                 const uint32_t sy,
                 const uint32_t sb_cols,
                 const uint32_t rows,
                 const uint32_t cols,
                 uint8_t * buf,
                 const bool is_y,
                 const uint32_t step_max,
                 const uint32_t lvl_shift,
                 const uint32_t row_mult) {
  const uint32_t x = sx * blockDim.x;
  const uint32_t y = sy * blockDim.x + threadIdx.x;
  uint32_t step = 0;

  if (y >= rows) {
    return;
  }

  const LOOP_FILTER_MASK* const mask = &lfm[sy*sb_cols + sx];
  uint64_t mask_16x16, mask_8x8, mask_4x4, mask_int_4x4;
  if (is_y) {
    mask_16x16 = mask->left_y[TX_16X16];
    mask_8x8 = mask->left_y[TX_8X8];
    mask_4x4 = mask->left_y[TX_4X4];
    mask_int_4x4 = mask->int_4x4_y;
  } else {
    mask_16x16 = mask->left_uv[TX_16X16];
    mask_8x8 = mask->left_uv[TX_8X8];
    mask_4x4 = mask->left_uv[TX_4X4];
    mask_int_4x4 = mask->int_4x4_uv;
  }

  const uint32_t row = threadIdx.x / MI_SIZE;
  const uint32_t mi_offset = row * step_max;
  uint32_t lfl_offset = row * row_mult << lvl_shift;

  for (; x + step * MI_SIZE < cols && step < step_max; step++) {
    const uint32_t shift = mi_offset + step;
    const uint32_t x_off = x + step * MI_SIZE;
    uint8_t* s = buf + y * cols + x_off;
    uint32_t lfl;
    if (is_y) {
      lfl = mask->lfl_y[lfl_offset];
    } else {
      lfl = mask->lfl_uv[lfl_offset];
    }
    const cuda_loop_filter_thresh llft = lft[lfl];
    const uint8_t mblim = llft.mblim;
    const uint8_t lim = llft.lim;
    const uint8_t hev_thr = llft.hev_thr;
    const uint32_t apply_16x16 = ((mask_16x16 >> shift) & 1);
    const uint32_t apply_8x8 = ((mask_8x8 >> shift) & 1);
    const uint32_t apply_4x4 = ((mask_4x4 >> shift) & 1);
    const uint32_t apply_int_4x4 = ((mask_int_4x4 >> shift) & 1);

    if (apply_16x16 | apply_8x8 | apply_4x4) {
      filter_vertical_edge(s, mblim, lim, hev_thr, apply_16x16, apply_8x8);
    }
    if(apply_int_4x4) {
      filter_vertical_edge(s + 4, mblim, lim, hev_thr, 0, 0);
    }

    lfl_offset += 1;
  }
}

__forceinline__ __device__
void filter_rows(const LOOP_FILTER_MASK* const lfm,
                 const cuda_loop_filter_thresh* const lft,
                 const uint32_t sx,
                 const uint32_t sy,
                 const uint32_t sb_cols,
                 const uint32_t rows,
                 const uint32_t cols,
                 uint8_t * buf,
                 const bool is_y,
                 const uint32_t step_max,
                 const uint32_t lvl_shift,
                 const uint32_t row_mult) {
  const uint32_t x = sx * blockDim.x + threadIdx.x;
  const uint32_t y = sy * blockDim.x;
  uint32_t step = 0;

  if (x >= cols) {
    return;
  }

  const LOOP_FILTER_MASK* const mask = &lfm[sy*sb_cols + sx];
  uint64_t mask_16x16, mask_8x8, mask_4x4, mask_int_4x4;
  if (is_y) {
    mask_16x16 = mask->above_y[TX_16X16];
    mask_8x8 = mask->above_y[TX_8X8];
    mask_4x4 = mask->above_y[TX_4X4];
    mask_int_4x4 = mask->int_4x4_y;
  } else {
    mask_16x16 = mask->above_uv[TX_16X16];
    mask_8x8 = mask->above_uv[TX_8X8];
    mask_4x4 = mask->above_uv[TX_4X4];
    mask_int_4x4 = mask->int_4x4_uv;
  }

  const uint32_t mi_offset = threadIdx.x / MI_SIZE;
  for (; y + step * MI_SIZE < rows && step < step_max; step++) {
    const uint32_t shift = mi_offset + step * step_max;
    const uint32_t y_off = y + step * MI_SIZE;
    uint8_t * s = buf + y_off * cols + x;

    // Calculate loop filter threshold
    const uint32_t lfl_offset = (step * row_mult << lvl_shift) + mi_offset;
    uint32_t lfl;
    if(is_y) {
      lfl = mask->lfl_y[lfl_offset];
    } else {
      lfl = mask->lfl_uv[lfl_offset];
    }
    const cuda_loop_filter_thresh llft = lft[lfl];
    const uint8_t mblim = llft.mblim;
    const uint8_t lim = llft.lim;
    const uint8_t hev_thr = llft.hev_thr;

    if ((mask_16x16 >> shift) & 1) {
      vp9_mb_lpf_horizontal_edge_w_cuda(s, cols, mblim, lim, hev_thr);
    }
    else if ((mask_8x8 >> shift) & 1) {
      vp9_mbloop_filter_horizontal_edge_cuda(s, cols, mblim, lim, hev_thr);
    }
    else if ((mask_4x4 >> shift) & 1) {
      vp9_loop_filter_horizontal_edge_cuda(s, cols, mblim, lim, hev_thr);
    }
    if ((mask_int_4x4 >> shift) & 1 && (is_y || y_off + 4 < rows)) {
      vp9_loop_filter_horizontal_edge_cuda(s + 4 * cols, cols, mblim, lim, hev_thr);
    }
  }
}

__global__
void filter_all(const LOOP_FILTER_MASK* const lfm,
                const cuda_loop_filter_thresh* const lft,
                volatile int32_t *col_row_filtered,
                volatile int32_t *col_col_filtered,
                const int32_t sb_rows,
                const int32_t sb_cols,
                const uint32_t rows,
                const uint32_t cols,
                uint8_t * buf,
                const bool is_y,
                const uint32_t step_max,
                const uint32_t lvl_shift,
                const uint32_t row_mult) {
  const uint32_t b_idx = blockIdx.x;
  if (b_idx == 0) {
    for (int32_t c = 0; c < sb_cols; c++) {
      filter_cols(lfm, lft, c, b_idx, sb_cols, rows, cols, buf, is_y,
                  step_max, lvl_shift, row_mult);
      __syncthreads();
      col_col_filtered[b_idx] = c;

      filter_rows(lfm, lft, c, b_idx, sb_cols, rows, cols, buf, is_y,
                  step_max, lvl_shift, row_mult);
      __syncthreads();
      col_row_filtered[b_idx] = c;
    }
    col_row_filtered[b_idx] = sb_cols;
    col_col_filtered[b_idx] = sb_cols;
  }
  else {
    for (int32_t c = 0; c < sb_cols; c++) {
      while(col_row_filtered[b_idx - 1] < c) {}
      filter_cols(lfm, lft, c, b_idx, sb_cols, rows, cols, buf, is_y,
                  step_max, lvl_shift, row_mult);
      while(col_col_filtered[b_idx - 1] < c + 1) {}
      __syncthreads();
      col_col_filtered[b_idx] = c;
      filter_rows(lfm, lft, c, b_idx, sb_cols, rows, cols, buf, is_y,
                  step_max, lvl_shift, row_mult);
      __syncthreads();
      col_row_filtered[b_idx] = c;
    }
    col_row_filtered[b_idx] = sb_cols;
    col_col_filtered[b_idx] = sb_cols;
  }
}

extern "C" {

/*
 * cuda_loopfilter takes a fully decode frame, buf, as well as an array of
 * masks, and the frame dimensions.  buf is assumed to be sizeof(rows * cols)
 */
void cuda_loopfilter(const LOOP_FILTER_MASK* lfm,
                     const cuda_loop_filter_thresh* lft,
                     uint32_t y_rows,
                     uint32_t y_cols,
                     uint8_t * y_buf,
                     uint32_t uv_rows,
                     uint32_t uv_cols,
                     uint8_t * u_buf,
                     uint8_t * v_buf) {
  unsigned char    *d_y_buf;
  unsigned char    *d_u_buf;
  unsigned char    *d_v_buf;
  int32_t * d_col_col_y_filtered,* d_col_col_u_filtered, *d_col_col_v_filtered;
  int32_t * d_col_row_y_filtered,* d_col_row_u_filtered, *d_col_row_v_filtered;

  static double total_time = 0;
  LOOP_FILTER_MASK* d_masks;
  cuda_loop_filter_thresh* d_lft;
  uint32_t y_buf_size = y_rows * y_cols * sizeof(uint8_t);
  uint32_t uv_buf_size = uv_rows * uv_cols * sizeof(uint8_t);
  hipStream_t y_s, u_s, v_s, gen1, gen2;
  hipStreamCreate(&y_s);
  hipStreamCreate(&u_s);
  hipStreamCreate(&v_s);
  hipStreamCreate(&gen1);
  hipStreamCreate(&gen2);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  gpuErrchk(hipMalloc( (void**)&d_y_buf, y_buf_size));
  gpuErrchk(hipMemcpyAsync(d_y_buf, (void*)y_buf, y_buf_size,
    hipMemcpyHostToDevice, y_s));

  gpuErrchk(hipMalloc( (void**)&d_u_buf, uv_buf_size));
  gpuErrchk(hipMemcpyAsync(d_u_buf, (void*)u_buf, uv_buf_size,
    hipMemcpyHostToDevice, u_s));

  gpuErrchk(hipMalloc( (void**)&d_v_buf, uv_buf_size));
  gpuErrchk(hipMemcpyAsync(d_v_buf, (void*)v_buf, uv_buf_size,
    hipMemcpyHostToDevice, v_s));

  gpuErrchk(hipMalloc( (void**)&d_lft,
    (MAX_LOOP_FILTER + 1) * sizeof(cuda_loop_filter_thresh)));
  gpuErrchk(hipMemcpyAsync(d_lft, (void*)lft,
    (MAX_LOOP_FILTER + 1) * sizeof(cuda_loop_filter_thresh),
    hipMemcpyHostToDevice, gen1));

  uint32_t sb_rows = (y_rows + SUPER_BLOCK_DIM - 1) / SUPER_BLOCK_DIM;
  uint32_t sb_cols = (y_cols + SUPER_BLOCK_DIM - 1) / SUPER_BLOCK_DIM;
  uint32_t sb_count = sb_rows * sb_cols;
  uint32_t mask_bytes = sb_count * sizeof(LOOP_FILTER_MASK);
  gpuErrchk(hipMalloc(&d_masks, mask_bytes));
  gpuErrchk(hipMemcpyAsync(d_masks, (void*)lfm, mask_bytes,
    hipMemcpyHostToDevice, gen2));

  gpuErrchk(hipMalloc( (void**)&d_col_row_y_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMalloc( (void**)&d_col_col_y_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMemsetAsync(d_col_row_y_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), y_s))
  gpuErrchk(hipMemsetAsync(d_col_col_y_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), y_s))

  gpuErrchk(hipMalloc( (void**)&d_col_row_u_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMalloc( (void**)&d_col_col_u_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMemsetAsync(d_col_row_u_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), u_s))
  gpuErrchk(hipMemsetAsync(d_col_col_u_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), u_s))

  gpuErrchk(hipMalloc( (void**)&d_col_row_v_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMalloc( (void**)&d_col_col_v_filtered,
    sb_rows * sizeof(int32_t)));
  gpuErrchk(hipMemsetAsync(d_col_row_v_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), v_s))
  gpuErrchk(hipMemsetAsync(d_col_col_v_filtered, 0xFFFFFFFF,
    sb_rows * sizeof(int32_t), v_s))
  // iterate over all diagonals

  dim3 work_block(SUPER_BLOCK_DIM);
  dim3 uv_work_block(SUPER_BLOCK_DIM / 2);
  dim3 work_grid(sb_rows);
  // First filter all left most columns on the diagonal frontier
  // These are the left most columns for each super block
  filter_all<<<work_grid, work_block,0,y_s>>>(d_masks, d_lft,
    d_col_row_y_filtered, d_col_col_y_filtered, sb_rows, sb_cols, y_rows,
    y_cols, d_y_buf, true, MI_SIZE, Y_LFL_SHIFT, 1);
  filter_all<<<work_grid, uv_work_block,0,u_s>>>(d_masks, d_lft,
    d_col_row_u_filtered, d_col_col_u_filtered, sb_rows, sb_cols, uv_rows,
    uv_cols, d_u_buf, false, MI_SIZE / 2, UV_LFL_SHIFT, 2);
  filter_all<<<work_grid, uv_work_block,0,v_s>>>(d_masks, d_lft,
    d_col_row_v_filtered, d_col_col_v_filtered, sb_rows, sb_cols, uv_rows,
    uv_cols, d_v_buf, false, MI_SIZE / 2, UV_LFL_SHIFT, 2);

  gpuErrchk(hipMemcpyAsync(y_buf, d_y_buf, y_buf_size,
    hipMemcpyDeviceToHost, y_s));

  gpuErrchk(hipMemcpyAsync(u_buf, d_u_buf, uv_buf_size,
    hipMemcpyDeviceToHost, u_s));
  gpuErrchk(hipMemcpyAsync(v_buf, d_v_buf, uv_buf_size,
    hipMemcpyDeviceToHost, v_s) );
  hipStreamDestroy(y_s);
  hipStreamDestroy(u_s);
  hipStreamDestroy(v_s);
  hipStreamDestroy(gen1);
  hipStreamDestroy(gen2);
  hipFree(d_y_buf);
  hipFree(d_u_buf);
  hipFree(d_v_buf);
  hipFree(d_masks);
  hipFree(d_lft);
  hipFree(d_col_col_y_filtered);
  hipFree(d_col_row_y_filtered);
  hipFree(d_col_col_u_filtered);
  hipFree(d_col_row_u_filtered);
  hipFree(d_col_col_v_filtered);
  hipFree(d_col_row_v_filtered);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float gpu_time;
  hipEventElapsedTime(&gpu_time, start, stop);
  total_time += gpu_time;
  printf("Elapsed GPU timer: %dX%d %.5fms, total_time %.5fms\n", y_cols,
    y_rows, gpu_time, total_time);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
}

